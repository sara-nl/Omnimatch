/**************************************************************************
 * Copyright (C) 2010 Pieter van Beek
 *
 * This file is part of SARAFFT.
 *
 * SARAFFT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * SARAFFT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with SARAFFT.  If not, see <http://www.gnu.org/licenses/>.
 **************************************************************************/

#include "omnicuda.h"

struct PlanList {
  struct PlanList *next;
  hipfftHandle plan;
  size_t size;
};

static struct PlanList *planList = NULL;


size_t getPlanSize( hipfftHandle plan ) {
  PlanList *current = planList;
  while ( current )
    if ( current->plan == plan )
      return current->size;
    else current = current->next;
  return 0;
}


bool destroyPlanSize( hipfftHandle plan ) {
  PlanList **current = &planList;
  while ( *current )
    if ( ( *current )->plan == plan ) {
      PlanList *next = ( *current )->next;
      free( ( void* )( *current ) );
      *current = next;
      return true;
    } else current = &( ( *current )->next );
  return false;
}


void setPlanSize( hipfftHandle plan, size_t size ) {
  destroyPlanSize( plan );
  PlanList *record = (PlanList*)malloc( sizeof( PlanList ) );
  record->plan = plan;
  record->size = size;
  record->next = planList;
  planList = record;
}


sararfftnd_plan sararfft3d_create_plan(
  int nx, int ny, int nz, sarafft_direction dir
) {
  sararfftnd_plan plan;
  hipfftResult result = hipfftPlan3d( &plan, nx, ny, nz, dir );
  if ( HIPFFT_SUCCESS != result )
    exit( -1 ); // TODO better error handling (but to do that, the caller must be rewritten)
  setPlanSize ( plan, sizeof( sarafft_real ) * nx * ny * nz );
  return plan;
}


void sararfftnd_destroy_plan(
  sararfftnd_plan plan
) {
  hipfftDestroy(plan);
  destroyPlanSize(plan);
}


void sararfftnd_one_real_to_complex(
  sararfftnd_plan p, sarafft_real *data, sarafft_complex *out
) {
  hipfftResult result = hipfftExecR2C( p, data, (hipfftComplex*)data );
}


void sararfftnd_one_complex_to_real(
  sararfftnd_plan p, sarafft_complex *in, sarafft_real    *out
) {

}


